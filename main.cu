#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "kernels.cu"
#include "utils.cu"

#include <iostream>
#include <fstream>

// Measure a more-realistic optimal bandwidth by a simple, memcpy-like kernel 
int bandwidthMemcpy( const uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                   , const size_t   N     // length of the input array
                   , int* d_in            // device input  of length N
                   , int* d_out           // device result of length N
) {
    // dry run to exercise the d_out allocation!
    const uint32_t num_blocks = (N + B - 1) / B;
    naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    { // timing the naivememcpy 
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            naiveMemcpy<<< num_blocks, B >>>(d_out, d_in, N);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int) * 1.0e-3f / elapsed;
        printf("Naive Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"
              , elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
    return 0;
}

// Measure bandwith of Cuda Memcpy device to device
int bandwidthCudaMemcpy( const size_t   N     // length of the input array
                   , int* d_in            // device input  of length N
                   , int* d_out           // device result of length N
) {

    double gigaBytesPerSec;
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;

    {
        gettimeofday(&t_start, NULL); 

        for(int i=0; i<RUNS_GPU; i++) {
            hipMemcpy(d_out, d_in, N, hipMemcpyDeviceToDevice);
        }
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
        gigaBytesPerSec = 2 * N * sizeof(int) * 1.0e-3f / elapsed;
        printf("Cuda Memcpy GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n\n"
              , elapsed, gigaBytesPerSec);
    }
 
    gpuAssert( hipPeekAtLastError() );
    return 0;
}

// Function that benchmark and validate the single pass scan 
// Return the gigaBytesPerSec of the sps 
template<class OP>
int spScanInc( uint32_t B     // desired CUDA block size ( <= 1024, multiple of 32)
                   , uint32_t N     // length of the input array
                   , int* h_in            // host input    of size: N * sizeof(int)
                   , int* d_in            // device input  of size: N * sizeof(int)
                   , int* d_out           // device result of size: N * sizeof(int)
                   , uint32_t kernel_version    // scan kernel version
                   , uint32_t chunk 
                   , int validate     
) {

    const size_t mem_size = N * sizeof(int);
    int* h_out = (int*)malloc(mem_size);
    int* h_ref = (int*)malloc(mem_size);
    hipMemset(d_out, 0, N*sizeof(int));

    // kernel parameters 
    const uint32_t CHUNK = chunk;
    const uint32_t elems_per_block = B * CHUNK;
    const uint32_t num_blocks = (N + elems_per_block - 1) / elems_per_block;
    const uint32_t shared_mem_size = B * sizeof(typename OP::ElTp) * CHUNK;
    printf("elems_per_block=%d, CHUNK=%d, num_blocks=%d, shmem_size=%d\n", elems_per_block, CHUNK, num_blocks, shared_mem_size);

    // mallocs 
    typename OP::ElTp* aggregates;
    typename OP::ElTp* prefixes;
    uint8_t* flags;
    uint32_t* dyn_block_id;

    hipMalloc((void**)&aggregates, num_blocks*sizeof( typename OP::ElTp));
    hipMalloc((void**)&prefixes, num_blocks*sizeof( typename OP::ElTp));
    hipMalloc((void**)&flags, num_blocks*sizeof( uint8_t ));
    hipMalloc((void**)&dyn_block_id, sizeof( uint32_t ));

    // ------- 10 dry run to exercise d_tmp allocation ------- //
    for (int i=0; i<10; i++) {
        hipMemset(flags, INC, num_blocks * sizeof(uint8_t));
        hipMemset(dyn_block_id, 0, sizeof(uint32_t));
        // choose which version of the kernel to run
        switch (kernel_version)
        {
        case 0:
            spScanKernelDepr<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 1:
            spScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 2: 
            spLookbackScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 3: 
            spWarpLookbackScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        default:
            printf("Kernel Version must be a value between 0-3\n");
            printf("<kernel-version>:\n"
            "    - 0: Naive implementation that uses global memory (spScanKernelDepr)\n"
            "    - 1: Without loopback (spScanKernel)\n"
            "    - 2: Single thread Loopback (spLookbackScanKernel)\n"
            "    - 3: Warp Loopback (spWarpLookbackScanKernel)\n\n");            
            exit(1);
        }
    }
    // ------------------------------ //

    // ---------- Time the GPU computation---------- //
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL); 

    for(int i=0; i<RUNS_GPU; i++) {

        // reset before every execution
        hipMemset(flags, INC, num_blocks * sizeof(uint8_t));
        hipMemset(dyn_block_id, 0, sizeof(uint32_t));

        // choose which version of the kernel to run
        switch (kernel_version)
        {
        case 0:
            spScanKernelDepr<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 1:
            spScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 2: 
            spLookbackScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        case 3: 
            spWarpLookbackScanKernel<OP, CHUNK><<<num_blocks, B, shared_mem_size>>>(d_out, d_in, aggregates, prefixes, flags, dyn_block_id, N);
            break;
        default:
            printf("Kernel Version must be a value between 0-3\n");
            printf("<kernel-version>:\n"
            "    - 0: Naive implementation that uses global memory (spScanKernelDepr)\n"
            "    - 1: Without loopback (spScanKernel)\n"
            "    - 2: Single thread Loopback (spLookbackScanKernel)\n"
            "    - 3: Warp Loopback (spWarpLookbackScanKernel)\n\n");            
            exit(1);
        }
    }
    hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_GPU;
    double gigaBytesPerSec = N  * (2*sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
    printf("Single Pass Scan GPU Kernel runs in: %lu microsecs, GB/sec: %.2f\n"
          , elapsed, gigaBytesPerSec);

    gpuAssert( hipPeekAtLastError() );
    //-------------------------------------//

    // ---------- Sequential computation for validation --------------- //
    if (validate) { 
        gettimeofday(&t_start, NULL);
        // printf("INPUT:\n");
        for(int i=0; i<RUNS_CPU; i++) {
            int acc = 0;
            for(uint32_t i=0; i<N; i++) {
                acc += h_in[i];
                h_ref[i] = acc;
                // printf("%d ", h_in[i]);
            }
            // printf("\n\n");
        }
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / RUNS_CPU;
        double gigaBytesPerSec = N * (sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
        printf("Scan CPU Sequential runs in: %lu microsecs, GB/sec: %.2f\n"
              , elapsed, gigaBytesPerSec);
    
        hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

        // printf("REF OUTPUT\n");
        // for(uint32_t i = 0; i<N; i++) {
        //     printf("%d ", h_ref[i]);
        // }
        // printf("\n");

        // printf("SPS OUTPUT\n");
        // for(uint32_t i = 0; i<N; i++) {
        //     printf("%d ", h_out[i]);
        // }
        // printf("\n\n");

        for(uint32_t i = 0; i<N; i++) {
            if(h_out[i] != h_ref[i]) {
                printf("!!!INVALID!!!: Single Pass Scan at index %d, dev-val: %d, host-val: %d\n"
                      , i, h_out[i], h_ref[i]);
                exit(1);
            }
        }
        printf("Single Pass Scan: VALID result!\n\n");
        // --------------------------------------- //
    }

    free(h_out);
    free(h_ref);

    // return bandwidth of sps 
    return gigaBytesPerSec;
}

int main (int argc, char * argv[]) {

    if (argc != 5) {
        printf("Usage: %s <benchmark> <array-length> <block-size> <kernel-version>\n", argv[0]);
        printf("<kernel-version>:\n"
        "    - 0: Naive implementation that uses global memory (spScanKernelDepr)\n"
        "    - 1: Without loopback (spScanKernel)\n"
        "    - 2: Single thread Loopback (spLookbackScanKernel)\n"
        "    - 3: Warp Loopback (spWarpLookbackScanKernel)\n\n");
        exit(1);
    }

    initHwd();

    // Configuration from main call 
    const uint8_t BENCHMARK = atoi(argv[1]);
    const uint32_t N = atoi(argv[2]);
    const uint32_t B = atoi(argv[3]);
    const uint8_t KERNEL = atoi(argv[4]);
    const uint32_t CHUNK = 12;

    int* h_in;
    int* d_in;
    int* d_out;

    if (BENCHMARK) {
        
        // Try different configuration
        const uint32_t kernel_versions[] = {2,3};
        const uint32_t n_sizes[] = {1024, 221184, 1000000, 10000000, 100003565}; 
        const uint32_t block_sizes[] = {128,256,512,1024};
        const uint32_t chunk_values[] = {1,2,6,10,12,14};

        int count = 0;

        std::ofstream results;
        results.open("benchmarks-sps.csv");
        results << "kernel,input,block,chunk,bandwidth\n";

        for (int kernel = 0; kernel < arrayLength(kernel_versions); kernel++) {
            for (int n = 0; n < arrayLength(n_sizes); n++) {
                for (int block_size = 0; block_size < arrayLength(block_sizes); block_size++) {
                    for (int c = 0; c < arrayLength(chunk_values); c++) {
                        
                        // write config of first run
                        results << kernel_versions[kernel] << "," << n_sizes[n] << "," << block_sizes[block_size] << "," << chunk_values[c] << ",";

                        count++;
                        printf("======== Bench Run %d =======\n", count);
                        printf("Configuration: KERNEL=%d, N=%d, B=%d, CHUNK=%d\n", kernel_versions[kernel], n_sizes[n], block_sizes[block_size], chunk_values[c]);
                        if (kernel_versions[kernel] == 3) printf("Latest Version of the SPScan Kernel is running...\n\n");
                        else printf("An older version of the SPScan Kernel is running. For the best performance run %s <array-length> <block-size> 3\n\n", argv[0]);

                        // run with current config 
                        const size_t mem_size = n_sizes[n]*sizeof(int);
                        h_in = (int*) malloc(mem_size);
                        hipMalloc((void**)&d_in ,   mem_size);
                        hipMalloc((void**)&d_out,   mem_size);

                        initArray(h_in, n_sizes[n], 13);
        
                        // run the single pass scan 
                        //double gigaBytesPerSec = spScanInc<Add<int>>(block_sizes[block_size], n_sizes[n], h_in, d_in, d_out, kernel_versions[kernel], chunk_values[c], 0);

                        // write result
                        //results << gigaBytesPerSec << "\n";

                    }
                }
            }
        }
        results.close();

    } else {

        // Check parameters
        if (B % 32 != 0) {
            printf("Block size must be a multiple of 32!\n");
            exit(1);
        }
        if (KERNEL > 3) {
            printf("Kernel version must be one between 0-3\n");
        }

        // Info Current Run
        printf("N=%d, B=%d, Kernel Version=%d\n", N, B, KERNEL);
        if (KERNEL == 3) printf("Latest Version of the SPScan Kernel is running...\n\n");
        else printf("An older version of the SPScan Kernel is running. For the best performance run %s <array-length> <block-size> 3\n\n", argv[0]);

        const size_t mem_size = N*sizeof(int); 
        h_in = (int*) malloc(mem_size);
        hipMalloc((void**)&d_in ,   mem_size);
        hipMalloc((void**)&d_out,   mem_size);

        initArray(h_in, N, 13);
        hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

        // computing a "realistic/achievable" bandwidth figure
        bandwidthMemcpy(B, N, d_in, d_out);
        
        // Cuda memcpy bandwidth
        bandwidthCudaMemcpy(mem_size, d_in, d_out);
        
        // run the single pass scan 
        spScanInc<Add<int>>(B, N, h_in, d_in, d_out, KERNEL, CHUNK, 1);

    }

    // cleanup memory
    free(h_in);
    hipFree(d_in );
    hipFree(d_out);

}
