#include "hip/hip_runtime.h"
#ifndef SCAN_HOST
#define SCAN_HOST

#include "utils.cu"
#include "kernels.cu"

uint32_t closestMul32(uint32_t x) {
    return ((x + 31) / 32) * 32;
}

void log2UB(uint32_t n, uint32_t* ub, uint32_t* lg) {
    uint32_t r = 0;
    uint32_t m = 1;
    if( n <= 0 ) { printf("Error: log2(0) undefined. Exiting!!!"); exit(1); }
    while(m<n) {
        r = r + 1;
        m = m * 2;
    }
    *ub = m;
    *lg = r;
}

/**
 * Host Wrapper orchestraiting the execution of scan:
 * d_in  is the input array
 * d_out is the result array (result of scan)
 * t_tmp is a temporary array (used to scan in-place across the per-block results)
 * Implementation consist of one phase
 */
template<class OP>                     // element-type and associative operator properties
void scanInc( const uint32_t     B     // desired CUDA block size ( <= 1024, multiple of 32)
            , const size_t       N     // length of the input array
            , typename OP::ElTp* d_out
            , typename OP::ElTp* d_in
            ){

    const uint32_t CHUNK = ELEMS_PER_THREAD*4 / sizeof(typename OP::ElTp);
    uint32_t num_seq_chunks;
    const uint32_t num_blocks = (N + B - 1) / B;
    const size_t   shmem_size = B * sizeof(typename OP::ElTp) * CHUNK;

    typename OP::ElTp* aggregs;
    typename OP::ElTp* prefs;
    char* flags;

    hipMalloc((void**)&aggregs, num_blocks*sizeof( typename OP::ElTp));
    hipMalloc((void**)&prefs, num_blocks*sizeof( typename OP::ElTp));
    hipMalloc((void**)&flags, num_blocks*sizeof( char ));


    scan3rdKernel<OP, CHUNK><<< num_blocks, B, shmem_size >>>(d_out, d_in, aggregs, prefs, flags, N);
}

#endif